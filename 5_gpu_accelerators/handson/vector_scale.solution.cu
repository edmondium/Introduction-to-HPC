
#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>



__global__ void vector_scale(float * vector, int count, float scalar)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < count)
    {
        vector[idx] *= scalar;
    }
}



int main()
{
    int count = 56789;
    int count_print = std::min(count, 10);
    float scalar = 10;

    float * h_vector = new float[count];
    for(int i = 0; i < count; i++) h_vector[i] = i;

    printf("Input: ");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_vector[i]);
    printf("\n");



    float * d_vector;
    hipMalloc(&d_vector, count * sizeof(float));

    hipMemcpy(d_vector, h_vector, count * sizeof(float), hipMemcpyHostToDevice);

    int tpb = 256;
    int bpg = (count - 1) / tpb + 1;
    vector_scale<<< bpg, tpb >>>(d_vector, count, scalar);

    hipMemcpy(h_vector, d_vector, count * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_vector);



    printf("Output:");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_vector[i]);
    printf("\n");

    int errors = 0;
    for(int i = 0; i < count; i++)
    {
        float correct = i * scalar;
        float observed = h_vector[i];
        if(std::abs((correct - observed) / correct) > 1e-4)
        {
            errors++;
            if(errors <= 5) printf("Wrong result on index %d: correct is %f, but result is %f\n", i, correct, observed);
        }
    }
    if(errors == 0) printf("Correct!\n");
    else printf("Total errors: %d\n", errors);

    delete[] h_vector;

    return 0;
}
