
#include <hip/hip_runtime.h>
#include <cstdio>



// TODO: write the say_hello kernel
__global__ void say_hello()
{
    int global_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;

    printf("Hello world from thread %d/%d, block %d/%d, my global index is %d/%d\n", threadIdx.x, blockDim.x, blockIdx.x, gridDim.x, global_idx, total_threads);
}



int main()
{
    // TODO: launch the say_hello kernel
    say_hello<<< 2, 4 >>>();

    // TODO: wait for the kernel to finish -- synchronize with the device
    hipDeviceSynchronize();

    return 0;
}
