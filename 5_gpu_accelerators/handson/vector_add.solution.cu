
#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>



// TODO: write the vector_add kernel
__global__ void vector_add(float * a, float * b, float * c, int count)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if(idx < count)
    {
        c[idx] = a[idx] + b[idx];
    }
}



int main()
{
    int count = 1234567;
    int count_print = std::min(count, 10);

    float * h_a = new float[count];
    float * h_b = new float[count];
    float * h_c = new float[count];
    for(int i = 0; i < count; i++) h_a[i] = i;
    for(int i = 0; i < count; i++) h_b[i] = 10 * i;

    printf("A:");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_a[i]);
    printf("\n");

    printf("B:");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_b[i]);
    printf("\n");



    // TODO: allocate GPU memory
    float * d_a;
    float * d_b;
    float * d_c;
    hipMalloc(&d_a, count * sizeof(float));
    hipMalloc(&d_b, count * sizeof(float));
    hipMalloc(&d_c, count * sizeof(float));

    // TODO: copy vectors A and B to the device
    hipMemcpy(d_a, h_a, count * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, count * sizeof(float), hipMemcpyHostToDevice);

    // TODO: set the number of threds per block
    int tpb = 256;
    // TODO: compute the total number of blocks needed to cover all elements of the vectors
    int bpg = (count - 1) / tpb + 1;
    // TODO: launch the vector add kernel
    vector_add<<<bpg,tpb>>>(d_a, d_b, d_c, count);

    // TODO: copy the vector C from the device
    hipMemcpy(h_c, d_c, count * sizeof(float), hipMemcpyDeviceToHost);

    // TODO: release GPU memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);



    printf("C:");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_c[i]);
    printf("\n");

    int errors = 0;
    for(int i = 0; i < count; i++)
    {
        float correct = 11 * i;
        float result = h_c[i];
        if(std::abs((correct - result) / correct) > 1e-4)
        {
            errors++;
            if(errors <= 5) printf("Error on index %d: correct is %f, but result is %f\n", i, correct, h_c[i]);
        }
    }
    if(errors == 0) printf("Correct!\n");
    else printf("Total errors: %d\n", errors);

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
