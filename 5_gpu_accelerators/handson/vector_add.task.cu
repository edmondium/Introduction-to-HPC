
#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>



// TODO: write the vector_add kernel




int main()
{
    int count = 1234567;
    int count_print = std::min(count, 10);

    float * h_a = new float[count];
    float * h_b = new float[count];
    float * h_c = new float[count];
    for(int i = 0; i < count; i++) h_a[i] = i;
    for(int i = 0; i < count; i++) h_b[i] = 10 * i;

    printf("A:");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_a[i]);
    printf("\n");

    printf("B:");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_b[i]);
    printf("\n");



    // TODO: allocate GPU memory
    
    // TODO: copy vectors A and B to the device
    

    // TODO: set the number of threds per block
    
    // TODO: compute the total number of blocks needed to cover all elements of the vectors
    
    // TODO: launch the vector add kernel
    

    // TODO: copy the vector C from the device
    

    // TODO: release GPU memory
    



    printf("C:");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_c[i]);
    printf("\n");

    int errors = 0;
    for(int i = 0; i < count; i++)
    {
        float correct = 11 * i;
        float result = h_c[i];
        if(std::abs((correct - result) / correct) > 1e-4)
        {
            errors++;
            if(errors <= 5) printf("Error on index %d: correct is %f, but result is %f\n", i, correct, h_c[i]);
        }
    }
    if(errors == 0) printf("Correct!\n");
    else printf("Total errors: %d\n", errors);

    delete[] h_a;
    delete[] h_b;
    delete[] h_c;

    return 0;
}
