
#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>



int main()
{
    int count = 56789;
    int count_print = std::min(count, 10);
    float scalar = 10;

    float * h_vector = new float[count];
    for(int i = 0; i < count; i++) h_vector[i] = i;

    printf("Input: ");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_vector[i]);
    printf("\n");



    // TODO: do the scaling on the GPU
    for(int i = 0; i < count; i++)
    {
        h_vector[i] *= scalar;
    }



    printf("Output:");
    for(int i = 0; i < count_print; i++) printf(" %7.2f", h_vector[i]);
    printf("\n");

    int errors = 0;
    for(int i = 0; i < count; i++)
    {
        float correct = i * scalar;
        float observed = h_vector[i];
        if(std::abs((correct - observed) / correct) > 1e-4)
        {
            errors++;
            if(errors <= 5) printf("Wrong result on index %d: correct is %f, but result is %f\n", i, correct, observed);
        }
    }
    if(errors == 0) printf("Correct!\n");
    else printf("Total errors: %d\n", errors);

    delete[] h_vector;

    return 0;
}
